#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include "clustering_backends.hpp"
#include "coreset.hpp"
#include "utils.hpp"
#include <random>
#include <opencv2/core.hpp>
#include <iostream>

__global__ void assignPixelsKernel(
    const unsigned char* input,
    unsigned char* output,
    int width,
    int height,
    const float* centers,
    int k,
    float color_scale,
    float spatial_scale)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total = width * height;
    if (idx >= total) return;

    int r = idx / width;
    int c = idx % width;
    int offset = idx * 3;

    float x01 = float(c) / float(width);
    float y01 = float(r) / float(height);

    // Create feature vector - scale colors like sequential version does
    float f[5] = {
        (float)input[offset + 0] * color_scale,  // B (not R!)
        (float)input[offset + 1] * color_scale,  // G
        (float)input[offset + 2] * color_scale,  // R
        x01 * spatial_scale,
        y01 * spatial_scale
    };

    int bestIdx = 0;
    float bestDist2 = 1e20f;

    for (int ci = 0; ci < k; ++ci) {
        float d2 = 0.0f;
        for (int d = 0; d < 5; ++d) {
            float diff = f[d] - centers[ci * 5 + d];
            d2 += diff * diff;
        }
        if (d2 < bestDist2) {
            bestDist2 = d2;
            bestIdx = ci;
        }
    }

    // Write output - divide by color_scale like sequential version
    float inv_scale = 1.0f / fmaxf(1e-6f, color_scale);
    output[offset + 0] = (unsigned char)fminf(255.0f, centers[bestIdx * 5 + 0] * inv_scale); // B
    output[offset + 1] = (unsigned char)fminf(255.0f, centers[bestIdx * 5 + 1] * inv_scale); // G
    output[offset + 2] = (unsigned char)fminf(255.0f, centers[bestIdx * 5 + 2] * inv_scale); // R
}

cv::Mat segmentFrameWithKMeans_cuda(
    const cv::Mat& frame,
    int k,
    int sample_size,
    float color_scale,
    float spatial_scale)
{
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    std::cout << "CUDA devices: " << deviceCount << ", status: " << hipGetErrorString(err) << std::endl;

    if (deviceCount == 0) {
        std::cerr << "No CUDA devices found!" << std::endl;
        return frame.clone(); // Return original frame
    }

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    std::cout << "Using device: " << prop.name << std::endl;

    CV_Assert(frame.type() == CV_8UC3);

    // Compute cluster centers on host
    std::vector<cv::Vec<float, 5>> centers = computeKMeansCenters(frame, k, sample_size, color_scale, spatial_scale);

    std::cout << "=== DEBUG INFO ===" << std::endl;
    std::cout << "Frame: " << frame.cols << "x" << frame.rows << std::endl;
    std::cout << "First input pixel BGR: "
        << (int)frame.data[0] << ", "
        << (int)frame.data[1] << ", "
        << (int)frame.data[2] << std::endl;

    int rows = frame.rows;
    int cols = frame.cols;
    int totalPixels = rows * cols;
    cv::Mat out(frame.size(), frame.type());

    // Initialize output to something visible (for testing)
    out = cv::Scalar(255, 0, 0); // Blue - if you see this, kernel didn't run

    // Allocate device memory
    unsigned char* d_input;
    unsigned char* d_output;
    float* d_centers;

    err = hipMalloc(&d_input, totalPixels * 3 * sizeof(unsigned char));
    std::cout << "hipMalloc d_input: " << hipGetErrorString(err) << std::endl;

    err = hipMalloc(&d_output, totalPixels * 3 * sizeof(unsigned char));
    std::cout << "hipMalloc d_output: " << hipGetErrorString(err) << std::endl;

    err = hipMalloc(&d_centers, k * 5 * sizeof(float));
    std::cout << "hipMalloc d_centers: " << hipGetErrorString(err) << std::endl;

    // Copy input frame to device
    err = hipMemcpy(d_input, frame.data, totalPixels * 3 * sizeof(unsigned char), hipMemcpyHostToDevice);
    std::cout << "hipMemcpy input: " << hipGetErrorString(err) << std::endl;

    // Flatten centers array
    std::vector<float> flatCenters(k * 5);
    for (int i = 0; i < k; ++i)
        for (int d = 0; d < 5; ++d)
            flatCenters[i * 5 + d] = centers[i][d];

    err = hipMemcpy(d_centers, flatCenters.data(), k * 5 * sizeof(float), hipMemcpyHostToDevice);
    std::cout << "hipMemcpy centers: " << hipGetErrorString(err) << std::endl;

    // Launch kernel
    int threadsPerBlock = 256;
    int blocks = (totalPixels + threadsPerBlock - 1) / threadsPerBlock;

    std::cout << "Launching kernel: " << blocks << " blocks, " << threadsPerBlock << " threads" << std::endl;

    assignPixelsKernel << <blocks, threadsPerBlock >> > (d_input, d_output, cols, rows, d_centers, k, color_scale, spatial_scale);

    err = hipGetLastError();
    std::cout << "Kernel launch: " << hipGetErrorString(err) << std::endl;

    err = hipDeviceSynchronize();
    std::cout << "Kernel execution: " << hipGetErrorString(err) << std::endl;

    // Copy result back to host
    err = hipMemcpy(out.data, d_output, totalPixels * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost);
    std::cout << "hipMemcpy output: " << hipGetErrorString(err) << std::endl;

    std::cout << "First output pixel BGR: "
        << (int)out.data[0] << ", "
        << (int)out.data[1] << ", "
        << (int)out.data[2] << std::endl;
    std::cout << "==================" << std::endl;

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_centers);

    return out;
}